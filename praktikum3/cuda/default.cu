#include "hip/hip_runtime.h"
#include "default.h"
#include <cstdio>

__global__ void matrix_mult(double *a, double *b, double *res) {
  for (int r = blockIdx.x * blockDim.x + threadIdx.x; r < SIZE; r += blockDim.x * gridDim.x)
    for (int c = blockIdx.y * blockDim.y + threadIdx.y; c < SIZE; c += blockDim.y * gridDim.y)
      for (int i = 0; i < SIZE; i++)
        res[r * SIZE + c] += a[r * SIZE + i] * b[i * SIZE + c];
}

void cuda_matrix_mult(double a[SIZE][SIZE], double b[SIZE][SIZE], double res[SIZE][SIZE]) {
  double *gpu_a;
  double *gpu_b;
  double *gpu_res;

  constexpr size_t byte_size = SIZE * SIZE * sizeof(double);

  hipError_t err = hipMallocManaged(&gpu_a, byte_size);
  if (err != hipSuccess)
    printf("malloc failed");
  err = hipMallocManaged(&gpu_b, byte_size);
  if (err != hipSuccess)
    printf("malloc failed");
  hipMallocManaged(&gpu_res, byte_size);
  if (err != hipSuccess)
    printf("malloc failed");

  for (int i = 0; i < SIZE * SIZE; i++) {
    gpu_a[i] = ((double*)a)[i];
    gpu_b[i] = ((double*)b)[i];
  }
  
  int deviceID;
  hipGetDevice(&deviceID);

  hipMemPrefetchAsync(gpu_a, byte_size, deviceID);
  hipMemPrefetchAsync(gpu_b, byte_size, deviceID);

  dim3 threadsPerBlock(1, 128);
  dim3 numBlocks(SIZE/(threadsPerBlock.x), SIZE/(threadsPerBlock.y));
  matrix_mult<<<numBlocks, threadsPerBlock>>>(gpu_a, gpu_b, gpu_res);

  hipDeviceSynchronize();

  hipMemPrefetchAsync(gpu_res, byte_size, hipCpuDeviceId);

  for (int i = 0; i < SIZE * SIZE; i++){
    ((double*)res)[i] = gpu_res[i];
  }

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_res);
}
