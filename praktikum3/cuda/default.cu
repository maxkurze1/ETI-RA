#include "hip/hip_runtime.h"
#include "default.h"
#include <cstdio>

__global__ void matrix_mult(double *a, double *b, double *res) {
  int r = blockIdx.x * blockDim.x + threadIdx.x;
  int c = blockIdx.y * blockDim.y + threadIdx.y;

  for (int i = 0; i < SIZE; i++)
    res[r * SIZE + c] += a[r * SIZE + i] * b[i * SIZE + c];
}

void cuda_matrix_mult(double a[SIZE][SIZE], double b[SIZE][SIZE], double res[SIZE][SIZE]) {
  double *gpu_a;
  double *gpu_b;
  double *gpu_res;

  constexpr size_t byte_size = SIZE * SIZE * sizeof(double);

  hipMalloc(&gpu_a, byte_size);
  hipMalloc(&gpu_b, byte_size);
  hipMalloc(&gpu_res, byte_size);

  hipMemcpy(gpu_a, a, byte_size, hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, b, byte_size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);
  matrix_mult<<<numBlocks, threadsPerBlock>>>(gpu_a, gpu_b, gpu_res);

  hipDeviceSynchronize();

  hipMemcpy(res, gpu_res, byte_size, hipMemcpyDeviceToHost);

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_res);
}
