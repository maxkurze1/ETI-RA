#include "hip/hip_runtime.h"
#include "default.h"
#include <cstdio>

__global__ void matrix_mult(double *a, double *b, double *res) {
  for (int r = blockIdx.x * blockDim.x + threadIdx.x; r < SIZE; r += blockDim.x * gridDim.x)
    for (int c = blockIdx.y * blockDim.y + threadIdx.y; c < SIZE; c += blockDim.y * gridDim.y)
      for (int i = 0; i < SIZE; i++)
        res[r * SIZE + c] += a[r * SIZE + i] * b[i * SIZE + c];
}

void cuda_matrix_mult(double a[SIZE][SIZE], double b[SIZE][SIZE], double res[SIZE][SIZE]) {
  double *gpu_a;
  double *gpu_b;
  double *gpu_res;

  constexpr size_t byte_size = SIZE * SIZE * sizeof(double);

  hipMallocManaged(&gpu_a, byte_size);
  hipMallocManaged(&gpu_b, byte_size);
  hipMallocManaged(&gpu_res, byte_size);

  // hipMemcpy(gpu_a, a, byte_size, hipMemcpyHostToDevice);
  // hipMemcpy(gpu_b, b, byte_size, hipMemcpyHostToDevice);
  for (int i = 0; i < SIZE * SIZE; i++) {
    gpu_a[i] = ((double*)a)[i];
    gpu_b[i] = ((double*)b)[i];
  }
  
  int deviceID;
  hipGetDevice(&deviceID);

  hipMemPrefetchAsync(gpu_a, byte_size, deviceID);
  hipMemPrefetchAsync(gpu_b, byte_size, deviceID);

  dim3 threadsPerBlock(1, 128);
  dim3 numBlocks(SIZE/(threadsPerBlock.x), SIZE/(threadsPerBlock.y));
  matrix_mult<<<numBlocks, threadsPerBlock>>>(gpu_a, gpu_b, gpu_res);

  hipDeviceSynchronize();

  hipMemPrefetchAsync(gpu_res, byte_size, deviceID);

  for (int i = 0; i < SIZE * SIZE; i++){
    ((double*)res)[i] = gpu_res[i];
  }
  // hipMemcpy(res, gpu_res, byte_size, hipMemcpyDeviceToHost);

  hipFree(gpu_a);
  hipFree(gpu_b);
  hipFree(gpu_res);
}
